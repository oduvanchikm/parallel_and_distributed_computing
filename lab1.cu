#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SIZE_BLOCK 512

__global__ void reverse_array_function(float* input_array, float* output_arra, int n)
{
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n; idx += gridDim.x * blockDim.x)
    {
        output_arra[n - 1 - idx] = input_array[idx];
    }
}

int main()
{
    int n;
    scanf("%d", &n);

    float* input_array = (float*)malloc(sizeof(float) * n);
    float* reverse_array = (float*)malloc(sizeof(float) * n);
    if (!input_array || !reverse_array)
    {
        printf("Allocation error in malloc\n");
        return 1;
    }

    for (int i = 0; i < n; ++i)
    {
        scanf("%f", &input_array[i]);
    }

    float* input_array_cuda;
    float* reverse_array_cuda;

    hipError_t error_1 = hipMalloc(&input_array_cuda, sizeof(float)* n);
    if (error_1 != hipSuccess)
    {
      printf("cudaMalloc1 failed: %s\n", hipGetErrorString(error_1));
      return 1;
    }


    hipError_t error_2 = hipMalloc(&reverse_array_cuda,sizeof(float)* n);
    if (error_2 != hipSuccess)
    {
      printf("Allocation error in cudaMalloc2");
      return 1;
    }

    hipError_t error_3 = hipMemcpy(input_array_cuda, input_array, n * sizeof(float), hipMemcpyHostToDevice);
    if (error_3 != hipSuccess)
    {
      printf("Allocation error in cudaMemcpy1");
      return 1;
    }

    reverse_array_function<<<SIZE_BLOCK, SIZE_BLOCK>>>(input_array_cuda, reverse_array_cuda, n);

    hipError_t error_4 = hipMemcpy(reverse_array, reverse_array_cuda,sizeof(float)* n, hipMemcpyDeviceToHost);
    if (error_4 != hipSuccess)
    {
      printf("Allocation error in cudaMemcpy2");
      return 1;
    }

    for (int i = 0; i < n; ++i)
    {
        printf("%.10e ", reverse_array[i]);
    }
    printf("\n");

    free(input_array);
    free(reverse_array);
    hipFree(input_array_cuda);
    hipFree(reverse_array_cuda);

    return 0;
}
